#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include "model.cuh"

#define THREAD_NUM 256

__global__ void scale_values(float *num, size_t size, float abs_max)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < size)
		num[idx] = (abs_max + abs_max) * num[idx] - abs_max;
}

void alloc_rand_values(float **dev_ptr, size_t size, hiprandGenerator_t *generator, float abs_max) {
    hipMalloc(dev_ptr, size);
    hiprandGenerateUniform(*generator, *dev_ptr, size);
    scale_values<<<(size + THREAD_NUM - 1) / THREAD_NUM, THREAD_NUM>>>(*dev_ptr, size, abs_max);
}

void alloc_zero_values(float **dev_ptr, size_t size) {
    hipMalloc(dev_ptr, size);
    hipMemset(*dev_ptr, 0, size);
}

SubModel::SubModel(SubModelSpec spec) {
    this->spec = spec;
/*
    // Initialize Curand generator
    hiprandGenerator_t randGen;
	hiprandCreateGenerator(&randGen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(randGen, (unsigned long long)clock());

    // Alloc weight matrices
    hipMalloc(&(this->weight_matrices), sizeof(float *) * spec.number_of_layers);

    alloc_rand_values(
        &(this->weight_matrices[0]),
        sizeof(float) * spec.number_of_input_nodes * spec.layers[0].number_of_nodes,
        &randGen,
        sqrt(6.0f / (spec.number_of_input_nodes + spec.layers[0].number_of_nodes))
    );

    for (int i = 1; i < spec.number_of_layers; i++) {
        alloc_rand_values(
            &(this->weight_matrices[i]),
            sizeof(float) * spec.layers[i - 1].number_of_nodes * spec.layers[i].number_of_nodes,
            &randGen,
            sqrt(6.0f / (spec.layers[i - 1].number_of_nodes + spec.layers[i].number_of_nodes))
        );
    }

    // Alloc biases & forward values
    hipMalloc(&(this->biases), sizeof(float *) * spec.number_of_layers);
    hipMalloc(&(this->forward_values), sizeof(float *) * spec.number_of_layers);
    hipMalloc(&(this->gradients), sizeof(float *) * spec.number_of_layers);

    for (int i = 0; i < spec.number_of_layers; i++) {
        alloc_zero_values(&(this->biases[i]), sizeof(float) * spec.layers[i].number_of_nodes);
        hipMalloc(&(this->forward_values[i]), sizeof(float) * spec.layers[i].number_of_nodes);
        hipMalloc(&(this->gradients[i]), sizeof(float) * spec.layers[i].number_of_nodes);
    }*/
}

SubModel::~SubModel() {/* 여기 오류있다 !
    for (int i = 0; i < spec.number_of_layers; i++) {
        hipFree(this->weight_matrices[i]);
        hipFree(this->biases[i]);
        hipFree(this->forward_values[i]);
        hipFree(this->gradients[i]);
    }

    hipFree(this->weight_matrices);
    hipFree(this->biases);
    hipFree(this->forward_values);
    hipFree(this->gradients);*/
}
