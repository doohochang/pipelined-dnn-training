#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <math.h>
#include "model.cuh"
#include <stdio.h>

#define THREAD_NUM 256

__global__ void scale_values(float *num, size_t size, float abs_max)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < size)
		num[idx] = (abs_max + abs_max) * num[idx] - abs_max;
}

void alloc_rand_values(float **dev_ptr, size_t size, hiprandGenerator_t *generator, float abs_max) {
    hipMalloc(dev_ptr, size);
    hiprandGenerateUniform(*generator, *dev_ptr, size);
    scale_values<<<(size + THREAD_NUM - 1) / THREAD_NUM, THREAD_NUM>>>(*dev_ptr, size, abs_max);
}

void alloc_zero_values(float **dev_ptr, size_t size) {
    hipMalloc(dev_ptr, size);
    hipMemset(*dev_ptr, 0, size);
}

SubModel::SubModel(SubModelSpec spec) {
    this->spec = spec;

    // Initialize Curand generator
    hiprandGenerator_t randGen;
	hiprandCreateGenerator(&randGen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(randGen, (unsigned long long)clock());

    // Alloc weight matrices
    //hipMalloc(&(this->weight_matrices), sizeof(float *) * spec.number_of_layers); //이렇게하면 alloc_rand에서 오류발생

    hipMalloc(&(this->weight_matrices), sizeof(float *) * spec.number_of_layers);
    this->weight_matrices_buffer = (float**)malloc(sizeof(float *) * spec.number_of_layers);

    alloc_rand_values(
        &(this->weight_matrices_buffer[0]),
        sizeof(float) * spec.number_of_input_nodes * spec.layers[0].number_of_nodes,
        &randGen,
        sqrt(6.0f / (spec.number_of_input_nodes + spec.layers[0].number_of_nodes))
    );
  
    for (int i = 1; i < spec.number_of_layers; i++) {
        alloc_rand_values(
            &(this->weight_matrices_buffer[i]),
            sizeof(float) * spec.layers[i - 1].number_of_nodes * spec.layers[i].number_of_nodes,
            &randGen,
            sqrt(6.0f / (spec.layers[i - 1].number_of_nodes + spec.layers[i].number_of_nodes))
        );
    }
    
    hipMemcpy(this->weight_matrices, this->weight_matrices_buffer, sizeof(float *) * spec.number_of_layers, hipMemcpyHostToDevice);

    // Alloc biases & forward values
    /* //이렇게하면 오류발생
    hipMalloc(&(this->biases), sizeof(float *) * spec.number_of_layers);
    hipMalloc(&(this->forward_values), sizeof(float *) * spec.number_of_layers);
    hipMalloc(&(this->gradients), sizeof(float *) * spec.number_of_layers);*/

    this->biases_buffer = (float**)malloc(sizeof(float *) * spec.number_of_layers);
    this->forward_values_buffer = (float**)malloc(sizeof(float *) * spec.number_of_layers);
    this->gradients_buffer = (float**)malloc(sizeof(float *) * spec.number_of_layers);
    
    hipMalloc(&(this->biases), sizeof(float *) * spec.number_of_layers);
    hipMalloc(&(this->forward_values), sizeof(float *) * spec.number_of_layers);
    hipMalloc(&(this->gradients), sizeof(float *) * spec.number_of_layers);
    
    for (int i = 0; i < spec.number_of_layers; i++) {
        alloc_zero_values(&(this->biases_buffer[i]), sizeof(float) * spec.layers[i].number_of_nodes);
        hipMalloc(&(this->forward_values_buffer[i]), sizeof(float) * spec.layers[i].number_of_nodes);
        hipMalloc(&(this->gradients_buffer[i]), sizeof(float) * spec.layers[i].number_of_nodes);
    }
    
    hipMemcpy(this->biases, this->biases_buffer, sizeof(float *) * spec.number_of_layers, hipMemcpyHostToDevice);
    hipMemcpy(this->forward_values, this->forward_values_buffer, sizeof(float *) * spec.number_of_layers, hipMemcpyHostToDevice);
    hipMemcpy(this->gradients, this->gradients_buffer, sizeof(float *) * spec.number_of_layers, hipMemcpyHostToDevice);
}

SubModel::~SubModel() {
    
    for (int i = 0; i < spec.number_of_layers; i++) {
        hipFree(this->weight_matrices_buffer[i]);
        hipFree(this->biases_buffer[i]);
        hipFree(this->forward_values_buffer[i]);
        hipFree(this->gradients_buffer[i]);
    }

    hipFree(this->weight_matrices);
    hipFree(this->biases);
    hipFree(this->forward_values);
    hipFree(this->gradients);
}
