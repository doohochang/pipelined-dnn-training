#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <math.h>
#include "model.cuh"
#include <stdio.h>

#define THREAD_NUM 256

__global__ void scale_values(float *num, size_t size, float abs_max)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < size)
		num[idx] = (abs_max + abs_max) * num[idx] - abs_max;
}

void alloc_rand_values(float **dev_ptr, size_t size, hiprandGenerator_t *generator, float abs_max) {
    hipMalloc(dev_ptr, size);
    hiprandGenerateUniform(*generator, *dev_ptr, size);
    scale_values<<<(size + THREAD_NUM - 1) / THREAD_NUM, THREAD_NUM>>>(*dev_ptr, size, abs_max);
}

void alloc_zero_values(float **dev_ptr, size_t size) {
    hipMalloc(dev_ptr, size);
    hipMemset(*dev_ptr, 0, size);
}

SubModel::SubModel(SubModelSpec spec, int batch_size) {
    this->spec = spec;

    // Initialize Curand generator
    hiprandGenerator_t randGen;
	hiprandCreateGenerator(&randGen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(randGen, (unsigned long long)clock());

    // Alloc weight matrices
    //hipMalloc(&(this->weight_matrices), sizeof(float *) * spec.number_of_layers); //이렇게하면 alloc_rand에서 오류발생
    
    this->weight_matrices_size = spec.number_of_input_nodes * spec.layers[0].number_of_nodes;
    
    this->biases_size = spec.layers[0].number_of_nodes;
    this->forward_values_size = batch_size * spec.layers[0].number_of_nodes;
    

    for(int i = 1; i < spec.number_of_layers; i++){
        this->weight_matrices_size += spec.layers[i-1].number_of_nodes * spec.layers[i].number_of_nodes;
        this->biases_size += spec.layers[i].number_of_nodes;
        this->forward_values_size += batch_size * spec.layers[i].number_of_nodes;
    }
    
    this->gradients_size = this->weight_matrices_size;
    
    alloc_rand_values(
        &(this->weight_matrices),
        sizeof(float) * this->weight_matrices_size,
        &randGen,
        sqrt(6.0f / (spec.layers[0].number_of_nodes + spec.layers[1].number_of_nodes))
    );

    hipMalloc(&(this->forward_values), sizeof(float) * this->forward_values_size);
    hipMalloc(&(this->gradients), sizeof(float) * this->gradients_size);
    
    alloc_zero_values(&(this->biases), sizeof(float) * this->biases_size);

}

SubModel::~SubModel() {
    hipFree(this->weight_matrices);
    hipFree(this->biases);
    hipFree(this->forward_values);
    hipFree(this->gradients);
}
