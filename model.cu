#include <hip/hip_runtime.h>
#include <hiprand.h>
#include "model.cuh"

SubModel::SubModel(SubModelSpec spec) {
    this->spec = spec;

    // Alloc weight matrices
    hipMalloc(&(this->weight_matrices), sizeof(float *) * spec.number_of_layers);

    hipMalloc(&(this->weight_matrices[0]), sizeof(float) * spec.number_of_input_nodes * spec.layers[0].number_of_nodes);

    for (int i = 1; i < spec.number_of_layers; i++) {
        hipMalloc(&(this->weight_matrices[i]), sizeof(float) * spec.layers[i - 1].number_of_nodes * spec.layers[i].number_of_nodes);
    }

    // Alloc biases & forward values
    hipMalloc(&(this->biases), sizeof(float *) * spec.number_of_layers);
    hipMalloc(&(this->forward_values), sizeof(float *) * spec.number_of_layers);

    for (int i = 0; i < spec.number_of_layers; i++) {
        hipMalloc(&(this->biases[i]), sizeof(float) * spec.layers[i].number_of_nodes);
        hipMalloc(&(this->forward_values[i]), sizeof(float) * spec.layers[i].number_of_nodes);
    }
}

SubModel::~SubModel() {
    for (int i = 0; i < spec.number_of_layers; i++) {
        hipFree(this->weight_matrices[i]);
        hipFree(this->biases[i]);
        hipFree(this->forward_values[i]);
    }

    hipFree(this->weight_matrices);
    hipFree(this->biases);
    hipFree(this->forward_values);
}
