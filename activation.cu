#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "activation.cuh"

__device__ __forceinline__ float sigmoid(float a) {
    return 1.0 / (1.0 + exp (-a));
}

__global__ void sigmoid_kernel(float *vec, int len) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < len) {
        vec[index] = sigmoid(vec[index]);
    }
}

__device__ __forceinline__ float relu(float a) {
    return a < 0 ? 0 : a;
}

__global__ void relu_kernel(float *vec, int len) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < len) {
        vec[index] = relu(vec[index]);
    }
}

