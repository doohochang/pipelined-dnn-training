#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>

#include "hparams.cuh"
#include "model.cuh"
#include "forward.cuh"
#include "backward.cuh"

#define TRAIN_CASE 55000
#define TEST_CASE 10000
#define NUM_DEVICE 1
#define THREAD_LEN 256
#define N_HIDDEN 2
#define D_INPUT 784
#define D_HIDDEN_1 1000
#define D_HIDDEN_2 10
#define D_OUTPUT 10
#define LEARNIG_RATE 0.01
#define BATCH_SIZE 100
#define MERGE_EPOCH 1
#define EPOCH 1000

int main(int argc, char** argv) {
    
    HiddenLayer* hiddenlayers = (HiddenLayer*)malloc(sizeof(HiddenLayer)* N_HIDDEN);

    hiddenlayers[0].number_of_nodes = D_HIDDEN_1;
    hiddenlayers[0].activation = ACTIVATION_RELU;

    hiddenlayers[1].number_of_nodes = D_HIDDEN_2;
    hiddenlayers[1].activation = ACTIVATION_LINEAR;

    OutputLayer outputlayer;
    outputlayer.number_of_nodes = D_OUTPUT;
    outputlayer.loss = LOSS_SOFTMAX_CROSS_ENTROPY;

    ModelSpec modelspec;
    modelspec.number_of_input_nodes = D_INPUT;
    modelspec.number_of_hidden_layers = N_HIDDEN;
    modelspec.hidden_layers = hiddenlayers;
    
    HyperParams hyperparmeters;
    hyperparmeters.number_of_devices = NUM_DEVICE;
    hyperparmeters.model_spec = modelspec;
    hyperparmeters.epoch = EPOCH;
    hyperparmeters.merge_period_epoch = MERGE_EPOCH;
    hyperparmeters.batch_size = BATCH_SIZE;
    hyperparmeters.learning_rate = LEARNIG_RATE;
    
    SubModelSpec submodelspec;
    submodelspec.number_of_layers = N_HIDDEN;
    submodelspec.number_of_input_nodes = D_INPUT;
    submodelspec.layers = hiddenlayers;
    
    SubModel submodel(submodelspec); ////여기서 오류발생
/*    
    hipSetDevice(0);
    srand(time(NULL));
    
    float *tfloat, *train_input, *test_input;
    int *tint, *train_label, *test_label;
    
    tfloat = (float*)malloc(sizeof(float) * D_INPUT * TRAIN_CASE);
    hipMalloc(&train_input, sizeof(float) * D_INPUT * TRAIN_CASE);
	hipMalloc(&test_input, sizeof(float) * D_INPUT * TEST_CASE);
    
    tint = (int*)malloc(sizeof(int) * TRAIN_CASE);
    hipMalloc(&train_label, sizeof(int) * TRAIN_CASE);
	hipMalloc(&test_label, sizeof(int) * TEST_CASE);
    
    FILE *train_image_path, *test_image_path;
	FILE *train_label_path, *test_label_path;

	train_image_path = fopen("./data/train_image.txt", "r");
	train_label_path = fopen("./data/train_label.txt", "r");
	test_image_path = fopen("./data/test_image.txt", "r");
	test_label_path = fopen("./data/test_label.txt", "r");
    
    int buffer_size = 0;
	for(int n = 0; n < TRAIN_CASE; n++)
	{
		for(int m = 0; m < D_INPUT; m++)
		{
			fscanf(train_image_path, "%f", &tfloat[buffer_size++]);
		}
	}
	
	//get train_label
	buffer_size = 0;
	for(int n = 0; n < TRAIN_CASE; n++)
	{
		fscanf(train_label_path, "%d", &tint[buffer_size++]);
	}
    
    float fshuffle[D_INPUT];
	int ishuffle;
	for(int n = 0; n < TRAIN_CASE; n++)
	{
		int idx = rand() % (TRAIN_CASE - n) + n;
	
		ishuffle = tint[idx];
		tint[idx] = tint[n];
		tint[n] = ishuffle;
			
		memcpy(fshuffle, &tfloat[idx*D_INPUT], sizeof(float) * D_INPUT);
		memcpy(&tfloat[idx*D_INPUT], &tfloat[n*D_INPUT], sizeof(float) * D_INPUT);
		memcpy(&tfloat[n*D_INPUT], fshuffle, sizeof(float) * D_INPUT);
	}
    
    hipMemcpy(train_input, tfloat, sizeof(float) * D_INPUT * TRAIN_CASE, hipMemcpyHostToDevice);
	hipMemcpy(train_label, tint, sizeof(int) * TRAIN_CASE, hipMemcpyHostToDevice);
    
    //get test_input
	buffer_size = 0;
	for(int n = 0; n < TEST_CASE; n++)
	{
		for(int m = 0; m < D_INPUT; m++)
		{
			fscanf(test_image_path, "%f", &tfloat[buffer_size++]);
		}
	}

	hipMemcpy(test_input, tfloat, sizeof(float) * D_INPUT * TEST_CASE, hipMemcpyHostToDevice);

	//get test_label
	buffer_size = 0;
	for(int n = 0; n < TRAIN_CASE; n++)
	{
		fscanf(test_label_path, "%d", &tint[buffer_size++]);
	}

	fclose(train_image_path);
	fclose(train_label_path);
	fclose(test_image_path);
	fclose(test_label_path);
    
    float *input;
	int *label;

	hipMalloc(&input, sizeof(float) * D_INPUT * BATCH_SIZE);
	hipMalloc(&label, sizeof(int) * BATCH_SIZE);
    
    hipStream_t stream;
	hipStreamCreate(&stream);
    
    float *one;
    float *zero;
    float *batch_size_buffer;
    float lr = LEARNIG_RATE;
    float *learning_rate;
    
    hipMalloc(&one, sizeof(float) * D_HIDDEN_1 * BATCH_SIZE);
    hipMemset(one, 1,  sizeof(float) * D_HIDDEN_1 * BATCH_SIZE);
    
	hipMalloc(&zero, sizeof(float) * D_HIDDEN_1 * BATCH_SIZE);
    hipMemset(zero, 0,  sizeof(float) * D_HIDDEN_1 * BATCH_SIZE);
    
    hipMalloc(&batch_size_buffer, sizeof(float) * BATCH_SIZE);
    hipMalloc(&learning_rate, sizeof(float));
    
    float * loss;
    hipMalloc(&loss, sizeof(float));
    
    hipMemcpyAsync(learning_rate, &lr, sizeof(float), hipMemcpyHostToDevice, stream);
    
    //start = clock();
	for(int epoch = 0; epoch < EPOCH; epoch++)
	{
		for(int n = 0; n < TRAIN_CASE/BATCH_SIZE; n++)
		{
			hipMemcpyAsync(input, &train_input[n*BATCH_SIZE*D_INPUT], sizeof(float) * BATCH_SIZE * D_INPUT, hipMemcpyHostToDevice, stream);
			hipMemcpyAsync(label, &train_label[n*BATCH_SIZE], sizeof(int) * BATCH_SIZE, hipMemcpyHostToDevice, stream);

            run_forward(&submodel, input, BATCH_SIZE, stream, one);
            run_output_layer(outputlayer, submodel.forward_values[1], BATCH_SIZE, label, loss, submodel.gradients[1], stream, one, batch_size_buffer);
            run_backward(&submodel, D_OUTPUT, submodel.forward_values[1], submodel.gradients[1], BATCH_SIZE, learning_rate, stream, one, zero);
            
		}

		//test
    }
    
*/    
    return 0;

}

