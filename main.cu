#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>

#include "hparams.cuh"
#include "model.cuh"
#include "forward.cuh"
#include "backward.cuh"

#define TRAIN_CASE 55000
#define TEST_CASE 10000
#define NUM_DEVICE 1
#define THREAD_LEN 256
#define N_HIDDEN 1
#define D_INPUT 784
#define D_HIDDEN 1000
#define D_OUTPUT 10
#define LEARNIG_RATE 0.01
#define BATCH_SIZE 100
#define MERGE_EPOCH 1
#define EPOCH 1000


int main(int argc, char** argv) {
    
    HiddenLayer hiddenlayer;
    hiddenlayer.number_of_nodes = D_HIDDEN;
    hiddenlayer.activation = ACTIVATION_RELU;

    OutputLayer outputlayer;
    outputlayer.number_of_nodes = D_OUTPUT;
    outputlayer.loss = LOSS_SOFTMAX_CROSS_ENTROPY;

    ModelSpec modelspec;
    modelspec.number_of_input_nodes = D_INPUT;
    modelspec.number_of_hidden_layers = N_HIDDEN;
    modelspec.hidden_layers = &hiddenlayer;
    
    HyperParams hyperparmeters;
    hyperparmeters.number_of_devices = NUM_DEVICE;
    hyperparmeters.model_spec = modelspec;
    hyperparmeters.epoch = D_HIDDEN;
    hyperparmeters.merge_period_epoch = MERGE_EPOCH;
    hyperparmeters.batch_size = BATCH_SIZE;
    hyperparmeters.learning_rate = LEARNIG_RATE;
    

    SubModelSpec submodelspec;
    submodelspec.number_of_layers = N_HIDDEN;
    submodelspec.number_of_input_nodes = D_INPUT;
    submodelspec.layers = &hiddenlayer;
    
    SubModel submodel(submodelspec);
    
    hipSetDevice(0);
    srand(time(NULL));
    float *tfloat, *train_input, *test_input;
    int *tint, *train_label, *test_label;
    
    tfloat = (float*)malloc(sizeof(float) * D_INPUT * TRAIN_CASE);
    hipMalloc(&train_input, sizeof(float) * D_INPUT * TRAIN_CASE);
	hipMalloc(&test_input, sizeof(float) * D_INPUT * TEST_CASE);
    
    tint = (int*)malloc(sizeof(int) * TRAIN_CASE);
    hipMalloc(&train_label, sizeof(int) * TRAIN_CASE);
	hipMalloc(&test_label, sizeof(int) * TEST_CASE);
    
    FILE *train_image_path, *test_image_path;
	FILE *train_label_path, *test_label_path;

	train_image_path = fopen("./data/train_image.txt", "r");
	train_label_path = fopen("./data/train_label.txt", "r");
	test_image_path = fopen("./data/test_image.txt", "r");
	test_label_path = fopen("./data/test_label.txt", "r");
    
    int buffer_size = 0;
	for(int n = 0; n < TRAIN_CASE; n++)
	{
		for(int m = 0; m < D_INPUT; m++)
		{
			fscanf(train_image_path, "%f", &tfloat[buffer_size++]);
		}
	}
	
	//get train_label
	buffer_size = 0;
	for(int n = 0; n < TRAIN_CASE; n++)
	{
		fscanf(train_label_path, "%d", &tint[buffer_size++]);
	}
    
    float fshuffle[D_INPUT];
	int ishuffle;
	for(int n = 0; n < TRAIN_CASE; n++)
	{
		int idx = rand() % (TRAIN_CASE - n) + n;
	
		ishuffle = tint[idx];
		tint[idx] = tint[n];
		tint[n] = ishuffle;
			
		memcpy(fshuffle, &tfloat[idx*D_INPUT], sizeof(float) * D_INPUT);
		memcpy(&tfloat[idx*D_INPUT], &tfloat[n*D_INPUT], sizeof(float) * D_INPUT);
		memcpy(&tfloat[n*D_INPUT], fshuffle, sizeof(float) * D_INPUT);
	}
    
    hipMemcpy(train_input, tfloat, sizeof(float) * D_INPUT * TRAIN_CASE, hipMemcpyHostToDevice);
	hipMemcpy(train_label, tint, sizeof(int) * TRAIN_CASE, hipMemcpyHostToDevice);
    
    //get test_input
	buffer_size = 0;
	for(int n = 0; n < TEST_CASE; n++)
	{
		for(int m = 0; m < D_INPUT; m++)
		{
			fscanf(test_image_path, "%f", &tfloat[buffer_size++]);
		}
	}

	hipMemcpy(test_input, tfloat, sizeof(float) * D_INPUT * TEST_CASE, hipMemcpyHostToDevice);

	//get test_label
	buffer_size = 0;
	for(int n = 0; n < TRAIN_CASE; n++)
	{
		fscanf(test_label_path, "%d", &tint[buffer_size++]);
	}

	fclose(train_image_path);
	fclose(train_label_path);
	fclose(test_image_path);
	fclose(test_label_path);
    
    float *input;
	int *label;

	hipMalloc(&input, sizeof(float) * D_INPUT * BATCH_SIZE);
	hipMalloc(&label, sizeof(int) * BATCH_SIZE);
    
    hipStream_t stream;
	hipStreamCreate(&stream);
    
    float *one;
    float *zero;
    float *batch_size_buffer;
    float lr = LEARNIG_RATE;
    float *learning_rate;
    
    hipMalloc(&one, sizeof(float) * D_HIDDEN * BATCH_SIZE);
    hipMemset(one, 1,  sizeof(float) * D_HIDDEN * BATCH_SIZE);
    
	hipMalloc(&zero, sizeof(float) * D_HIDDEN * BATCH_SIZE);
    hipMemset(zero, 0,  sizeof(float) * D_HIDDEN * BATCH_SIZE);
    
    hipMalloc(&batch_size_buffer, sizeof(float) * BATCH_SIZE);
    hipMalloc(&learning_rate, sizeof(float));
    
    hipMemcpyAsync(learning_rate, &lr, sizeof(float), hipMemcpyHostToDevice, stream);
    
    //start = clock();
	for(int epoch = 0; epoch < EPOCH; epoch++)
	{
		for(int n = 0; n < TRAIN_CASE/BATCH_SIZE; n++)
		{
			hipMemcpyAsync(input, &train_input[n*BATCH_SIZE*D_INPUT], sizeof(float) * BATCH_SIZE * D_INPUT, hipMemcpyHostToDevice, stream);
			hipMemcpyAsync(label, &train_label[n*BATCH_SIZE], sizeof(int) * BATCH_SIZE, hipMemcpyHostToDevice, stream);

            run_forward(&submodel, input, BATCH_SIZE, stream, one);
            run_output_layer(outputlayer, *submodel.forward_values, BATCH_SIZE, label, submodel.loss, *submodel.gradients, stream, one, batch_size_buffer);
            run_backward(&submodel, D_OUTPUT, *submodel.forward_values, *submodel.gradients, BATCH_SIZE, learning_rate, stream, one, zero);
            
		}

		//test
    }
    
    
    return 0;
}

