#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>

#include "hparams.cuh"
#include "forward.cuh"
#include "model.cuh"
#include "activation.cuh"

#define THREAD_NUM 256

__global__ void times(float *input, unsigned int input_size, float *output, unsigned int n) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if(index < n * input_size)
        output[index] = input[index % input_size];
}

void run_forward_step(
    hipblasHandle_t handle,
    hipStream_t stream,
    Activation activation,
    float *input, unsigned int batch_size, unsigned int input_size,
    float *weight_matrix, float *bias,
    float *output, unsigned int output_size,
    float *one
) {
    // weight_matrix: input_size * output_size
    // bias: output_size
 
    times<<<(batch_size * output_size + THREAD_NUM - 1) / THREAD_NUM, THREAD_NUM, 0, stream>>>(bias, output_size, output, batch_size);

    // input(batch_size, input_size) * weight_matrix(input_size, output_size) = output(batch_size, output_size)
    hipblasSgemm(
        handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
        batch_size, input_size, output_size,
        one,
        input, batch_size,
        weight_matrix, input_size,
        one,
        output, batch_size
    );

    switch (activation) {
        case ACTIVATION_LINEAR:
            break;
        case ACTIVATION_SIGMOID:
            sigmoid_kernel<<<(batch_size * output_size + THREAD_NUM - 1) / THREAD_NUM, THREAD_NUM, 0, stream>>>(output, batch_size * output_size);
            break;
        case ACTIVATION_RELU:
            relu_kernel<<<(batch_size * output_size + THREAD_NUM - 1) / THREAD_NUM, THREAD_NUM, 0, stream>>>(output, batch_size * output_size);
            break;
    }
}


__global__ void exp_kernel(float *array, unsigned int size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size)
		array[index] = exp(array[index]);
}

__global__ void set_value(float value, float *array, unsigned int size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size)
        array[index] = value;
}

__global__ void divide_by_vector(float *matrix, float *vector, unsigned int row, unsigned int col) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < row * col)
        matrix[index] /= vector[index / col];
}

__global__ void minus_one(float *matrix, int *indices, unsigned int row, unsigned int col) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < row)
        matrix[index * col + indices[index]] -= 1;
}

__global__ void pick_minus_log_ps(float *matrix, float *minus_log_ps, int *indices, unsigned int row, unsigned int col) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < row)
        minus_log_ps[index] = -log(matrix[index * col + indices[index]]);
}


void run_softmax_cross_entropy(float *scores, unsigned int batch_size, unsigned int number_of_scores, int *answers, float *loss, float *grad_scores, hipStream_t stream, hipblasHandle_t handle, const float *ones, float *batch_size_buffer) {
    
    hipMemcpyAsync(grad_scores, scores, sizeof(float) * batch_size * number_of_scores, hipMemcpyDeviceToDevice, stream);
    exp_kernel<<<(batch_size * number_of_scores + THREAD_NUM - 1) / THREAD_NUM, THREAD_NUM, 0, stream>>>(grad_scores, batch_size * number_of_scores);
    
    
    set_value<<<(batch_size + THREAD_NUM - 1) / THREAD_NUM, THREAD_NUM, 0, stream>>>(0, batch_size_buffer, batch_size);
    
    hipblasSgemv(
        handle, HIPBLAS_OP_N,
        batch_size, number_of_scores,
        ones,
        grad_scores, batch_size,
        ones, 1,
        ones,
        batch_size_buffer, 1
    );
    
    divide_by_vector<<<(batch_size * number_of_scores + THREAD_NUM - 1) / THREAD_NUM, THREAD_NUM, 0, stream>>>(grad_scores, batch_size_buffer, batch_size, number_of_scores);
    
    pick_minus_log_ps<<<(batch_size + THREAD_NUM - 1) / THREAD_NUM, THREAD_NUM, 0, stream>>>(grad_scores, batch_size_buffer, answers, batch_size, number_of_scores);
    
    hipblasSdot(handle, batch_size, batch_size_buffer, 1, ones, 1, loss);
    
    minus_one<<<(batch_size + THREAD_NUM - 1) / THREAD_NUM, THREAD_NUM, 0, stream>>>(grad_scores, answers, batch_size, number_of_scores);
    
    }

void run_output_layer(OutputLayer layer, float *input, unsigned int batch_size, int *answers, float *loss, float *grad_input, hipStream_t stream, hipblasHandle_t handle,  float *ones, float *batch_size_buffer, float* host_loss) {

    switch (layer.loss) {
        case LOSS_SOFTMAX_CROSS_ENTROPY:
            run_softmax_cross_entropy(input, batch_size, layer.number_of_input_nodes, answers, loss, grad_input, stream, handle, ones, batch_size_buffer);
            hipMemcpyAsync(host_loss, loss, sizeof(float), hipMemcpyDeviceToHost, stream);
            printf("%lf\n", *host_loss);
            break;
    }
}


void run_forward(SubModel *submodel, float *input, unsigned int batch_size, hipStream_t stream, float *one, OutputLayer outputlayer, float* loss, int* label, float *batch_size_buffer, float* host_loss) {
    
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetStream(handle, stream);
    
    int weight_matrices_start_index = 0;
    int biases_start_index = 0;
    int forward_values_start_index = 0;
    int forward_values_start_index_temp = 0;

    run_forward_step(
        handle, stream, submodel->spec.layers[0].activation,
        input, batch_size, submodel->spec.number_of_input_nodes,
        submodel->weight_matrices, submodel->biases,
        submodel->forward_values, submodel->spec.layers[0].number_of_nodes,
        one
    );
    
    for (int i = 1; i < submodel->spec.number_of_layers; i++){
        
        weight_matrices_start_index += submodel->spec.number_of_input_nodes * submodel->spec.layers[0].number_of_nodes;
        biases_start_index += batch_size * submodel->spec.layers[0].number_of_nodes;
        forward_values_start_index_temp = forward_values_start_index;
        forward_values_start_index += batch_size * submodel->spec.layers[0].number_of_nodes;

        run_forward_step(
            handle, stream, submodel->spec.layers[i].activation,
            submodel->forward_values + forward_values_start_index_temp, batch_size, submodel->spec.layers[i - 1].number_of_nodes,
            submodel->weight_matrices + weight_matrices_start_index, submodel->biases + biases_start_index,
            submodel->forward_values + forward_values_start_index, submodel->spec.layers[i].number_of_nodes,
            one
        );
    }
    
    if (submodel->spec.is_last_submodel){
        run_output_layer(outputlayer, submodel->forward_values + forward_values_start_index_temp,
                            batch_size, label, loss,
                            submodel->gradients + weight_matrices_start_index,
                            stream, handle, one, batch_size_buffer, host_loss);
        
    }
}



